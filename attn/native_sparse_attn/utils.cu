#include "hip/hip_runtime.h"
#include "utils.h"
#include <__clang_cuda_builtin_vars.h>
#include <assert.h>
#include <mma.h>

using namespace nvcuda;

__device__ inline float warpReduceSum(float val) {
  // Use warp shuffle down operations to perform reduction
  // Each step halves the number of active threads but combines values
  for (int offset = 16; offset > 0; offset /= 2) {
    val += __shfl_down_sync(0xffffffff, val, offset);
  }

  // At this point, lane 0 has the sum of all values in the warp
  return val;
}

__device__ float blockReduceSum(float val) {
  // Shared memory for storing warp sums
  static __shared__ float shared[32]; // One element per warp

  int lane = threadIdx.x % 32;   // Lane index within warp
  int warpId = threadIdx.x / 32; // Warp index within block

  // Perform warp reduction on each warp
  val = warpReduceSum(val);

  // Store the warp sum in shared memory (only lane 0 of each warp)
  if (lane == 0) {
    shared[warpId] = val;
  }

  // Make sure all warp sums are visible to all threads
  __syncthreads();

  // Read warp sums from shared memory (first warp only)
  val = (threadIdx.x < blockDim.x / 32) ? shared[lane] : 0;

  // Final reduction of warp sums (first warp only)
  if (warpId == 0) {
    val = warpReduceSum(val);
  }

  return val; // Result is stored in thread 0
}

template <int TILE_SIZE>
__device__ __inline__ void load_shared_tile(const __hip_bfloat16 *global_ptr, __hip_bfloat16 *shared_ptr,
                                            int global_stride, int shared_stride, int row_offset, int col_offset) {
#pragma unroll
  for (int i = threadIdx.y; i < TILE_SIZE; i += blockDim.y) {
#pragma unroll
    for (int j = threadIdx.x; j < TILE_SIZE; j += blockDim.x) {
      shared_ptr[i * shared_stride + j] = global_ptr[(row_offset + i) * global_stride + (col_offset + j)];
    }
  }
  __syncthreads();
}

__device__ float warpReduceMax(float val) {
  int laneId = threadIdx.x % warpSize;

  float warpMax = val;

  // Use warp shuffle down to compute max within each warp
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    float shflVal = __shfl_down_sync(0xffffffff, warpMax, offset);
    warpMax = max(warpMax, shflVal);
  }

  return warpMax;
}

template <int M, int N, int K>
__device__ __inline__ void bf16_warp_mm(const __hip_bfloat16 *matrix_a, // [M][K] column-major
                                        const __hip_bfloat16 *matrix_b, // [K][N] row-major
                                        float *matrix_c                // [M][N] row-major
) {
  assert(M % WMMA_M == 0);
  assert(N % WMMA_N == 0);

  int tid = threadIdx.x + threadIdx.y * blockDim.x;
  int warp_id = tid / warpSize;

  // tile output matrix
  int tiles_in_row = N / WMMA_N;
  int tiles_in_col = M / WMMA_M;

  // define warps tile
  int row_id = warp_id / tiles_in_row;
  int col_id = warp_id % tiles_in_row;

  // strides
  const int a_stride = M; // stride between cols
  const int b_stride = N; // stride between rows
  const int c_stride = N; // stride between rows

  // initial offsets
  int c_offset = row_id * (c_stride * WMMA_M) + col_id * WMMA_N;
  int a_offset = row_id * WMMA_M;
  int b_offset = col_id * WMMA_N;

  // declare wmma fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __hip_bfloat16, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __hip_bfloat16, wmma::row_major> b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  // fill output tile with 0
  wmma::fill_fragment(c_frag, 0.0f);

  // one thread per element in output matrix
  if (row_id < tiles_in_row && col_id < tiles_in_col) {
    // loop over inner dimension
#pragma unroll
    for (int k = 0; k < K; k += WMMA_K) {
      // load fragments
      wmma::load_matrix_sync(a_frag, matrix_a + a_offset + k * a_stride, a_stride);
      wmma::load_matrix_sync(b_frag, matrix_b + b_offset + k * b_stride, b_stride);

      // matmul accumulate on current tiles, C = AB + C
      wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    // store tile result to output matrix
    wmma::store_matrix_sync(matrix_c + c_offset, c_frag, c_stride, wmma::mem_row_major);
  }
}
