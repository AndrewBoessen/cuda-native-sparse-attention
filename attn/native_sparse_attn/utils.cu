#include "hip/hip_runtime.h"
#include "utils.h"
#include <assert.h>
#include <mma.h>

using namespace nvcuda;

__device__ inline float warpReduceSum(float val) {
  // Use warp shuffle down operations to perform reduction
  // Each step halves the number of active threads but combines values
  for (int offset = 16; offset > 0; offset /= 2) {
    val += __shfl_down_sync(0xffffffff, val, offset);
  }

  // At this point, lane 0 has the sum of all values in the warp
  return val;
}

__device__ float blockReduceSum(float val) {
  // Shared memory for storing warp sums
  static __shared__ float shared[32]; // One element per warp

  int lane = threadIdx.x % 32;   // Lane index within warp
  int warpId = threadIdx.x / 32; // Warp index within block

  // Perform warp reduction on each warp
  val = warpReduceSum(val);

  // Store the warp sum in shared memory (only lane 0 of each warp)
  if (lane == 0) {
    shared[warpId] = val;
  }

  // Make sure all warp sums are visible to all threads
  __syncthreads();

  // Read warp sums from shared memory (first warp only)
  val = (threadIdx.x < blockDim.x / 32) ? shared[lane] : 0;

  // Final reduction of warp sums (first warp only)
  if (warpId == 0) {
    val = warpReduceSum(val);
  }

  return val; // Result is stored in thread 0
}

template <int TILE_SIZE>
__device__ __inline__ void load_shared_tile(const __hip_bfloat16 *global_ptr, __hip_bfloat16 *shared_ptr,
                                            int global_stride, int shared_stride, int row_offset, int col_offset) {
#pragma unroll
  for (int i = threadIdx.y; i < TILE_SIZE; i += blockDim.y) {
#pragma unroll
    for (int j = threadIdx.x; j < TILE_SIZE; j += blockDim.x) {
      shared_ptr[i * shared_stride + j] = global_ptr[(row_offset + i) * global_stride + (col_offset + j)];
    }
  }
  __syncthreads();
}

template <int M, int N, int K>
__device__ __inline__ void bf16_warp_mm(const __hip_bfloat16 *matrix_a, // [M][K] column-major
                                        const __hip_bfloat16 *matrix_b, // [K][N] row-major
                                        float *matrix_c                // [M][N] row-major
) {
  // check if matrix fits in block
  assert(blockDim.x >= N);
  assert(blockDim.y >= M);

  // define warps tile
  int row_id = threadIdx.y / WMMA_M;
  int col_id = threadIdx.x / WMMA_N;

  // strides
  const int a_stride = M; // stride between cols
  const int b_stride = N; // stride between rows
  const int c_stride = N; // stride between rows

  // initial offsets
  int c_offset = row_id * (c_stride * WMMA_M) + col_id * WMMA_N;
  int a_offset = row_id * WMMA_M;
  int b_offset = col_id + WMMA_N;

  // declare wmma fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __hip_bfloat16, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, hip_bfloat16, wmma::row_major> b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  // fill output tile with 0
  wmma::fill_fragment(c_frag, 0.0f);

  // one thread per element in output matrix
  if (threadIdx.x < N && threadIdx.y < M) {
    // loop over inner dimension
#pragma unroll
    for (int k = 0; k < K; k += WMMA_K) {
      // load fragments
      wmma::load_matrix_sync(a_frag, matrix_a + a_offset + k * a_stride, a_stride);
      wmma::load_matrix_sync(b_frag, matrix_b + b_offset + k * b_stride, b_stride);

      // matmul accumulate on current tiles, C = AB + C
      wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    // store tile result to output matrix
    wmma::store_matrix_sync(matrix_c + c_offset, c_frag, c_stride, wmma::mem_row_major);
  }
}
