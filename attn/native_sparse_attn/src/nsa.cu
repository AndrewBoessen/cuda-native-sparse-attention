#include "hip/hip_runtime.h"
#include "../include/nsa.h"
#include "../include/utils.h"
#include <assert.h>

// Helper function to convert float to bfloat16
void convertFloatToBFloat16(const float *src, __hip_bfloat16 *dst, size_t size) {
  for (size_t i = 0; i < size; i++) {
    dst[i] = __float2bfloat16(src[i]);
  }
}

__global__ void mqa_kernel(const __hip_bfloat16 *query, const __hip_bfloat16 *key, const __hip_bfloat16 *value,
                           float *output, int seq_len, int num_heads, int head_dim, long **block_indices,
                           long *block_counts, int block_size, float scale_factor) {
  extern __shared__ __hip_bfloat16 smem[];

  // Offsets into shared memory
  __hip_bfloat16 *p_q = smem;
  __hip_bfloat16 *p_k = p_q + num_heads * head_dim;
  __hip_bfloat16 *p_v = p_k + block_size * head_dim;
  float *p_o = (float *)(p_v + block_size * head_dim);
  float *warp_reduce_scratch = p_o + num_heads * block_size;

  // Outer Loop (Q)
  int grid_row = blockIdx.y;
  const __hip_bfloat16 *q_bos = query + (grid_row * num_heads * head_dim);
  float *o_bos = output + (grid_row * num_heads * head_dim);

  // Inner Loop (KV)
  long num_blocks = block_counts[grid_row];
  for (int i = 0; i < num_blocks; i++) {
    long block_id = block_indices[grid_row][i];
    const __hip_bfloat16 *k_bos = key + (block_id * block_size * head_dim);
    const __hip_bfloat16 *v_bos = value + (block_id * block_size * head_dim);

    // Write blocks to shared memory
    int head_tiles = num_heads / warpSize;
    int dim_tiles = head_dim / warpSize;
    int block_tiles = block_size / warpSize;

#pragma unroll
    for (int j = 0; j < head_tiles; j += warpSize) {
#pragma unroll
      for (int k = 0; k < dim_tiles; k += warpSize) {
        load_shared_tile<__hip_bfloat16, MEM_TILE_SIZE>(q_bos, p_q, 1, head_dim, j, k);
        load_shared_tile<float, MEM_TILE_SIZE>(o_bos, p_o, 1, 1, j, k);
      }
    }
#pragma unroll
    for (int j = 0; j < block_tiles; j += warpSize) {
#pragma unroll
      for (int k = 0; k < dim_tiles; k += warpSize) {
        load_shared_tile<__hip_bfloat16, MEM_TILE_SIZE>(k_bos, p_k, 1, head_dim, j, k);
        load_shared_tile<__hip_bfloat16, MEM_TILE_SIZE>(v_bos, p_v, 1, 1, j, k);
      }
    }
  }
}

void launch_mqa_kernel(const __hip_bfloat16 *query, const __hip_bfloat16 *key, const __hip_bfloat16 *value, float *output,
                       int seq_len, int num_heads, int head_dim, long **block_indices, long *block_counts,
                       int block_size, float scale_factor, hipStream_t stream) {
  assert(num_heads % 16 == 0);
  assert(head_dim % 16 == 0);
  assert(block_size % 16 == 0);

  // Number of bytes in shared memory
  size_t qkv_mem_size = (num_heads * head_dim + 2 * (block_size * head_dim)) * sizeof(__hip_bfloat16);
  size_t output_tile_size = (num_heads * block_size) * sizeof(float);
  size_t warp_reduce_scratch_size = (block_size / 32 * num_heads) * sizeof(float);

  size_t sharedMem = qkv_mem_size + output_tile_size + warp_reduce_scratch_size;

  dim3 blockDim(block_size, num_heads);
  dim3 gridDim(seq_len);

  mqa_kernel<<<gridDim, blockDim, sharedMem, stream>>>(query, key, value, output, seq_len, num_heads, head_dim,
                                                       block_indices, block_counts, block_size, scale_factor);
}

void native_sparse_attention(const float *query, const float *key, const float *value, float *output, int batch_size,
                             int seq_len, int num_heads, int head_dim, long **block_indices, long *block_counts,
                             int block_size, float scale_factor) {
  // Number of elements in array
  size_t q_size = batch_size * seq_len * num_heads * head_dim;
  size_t kv_size = batch_size * seq_len * head_dim;
  size_t o_size = q_size;

  // Allocate memory on device
  __hip_bfloat16 *d_q, *d_k, *d_v;
  float *d_o;
  hipMalloc(&d_q, q_size * sizeof(__hip_bfloat16));
  hipMalloc(&d_k, kv_size * sizeof(__hip_bfloat16));
  hipMalloc(&d_v, kv_size * sizeof(__hip_bfloat16));
  hipMalloc(&d_o, o_size * sizeof(float));

  // Allocate host bfloat16 arrays
  __hip_bfloat16 *bf16_query = (__hip_bfloat16 *)malloc(q_size * sizeof(__hip_bfloat16));
  __hip_bfloat16 *bf16_key = (__hip_bfloat16 *)malloc(kv_size * sizeof(__hip_bfloat16));
  __hip_bfloat16 *bf16_value = (__hip_bfloat16 *)malloc(kv_size * sizeof(__hip_bfloat16));

  // Cast host float arrays to bfloat16
  convertFloatToBFloat16(query, bf16_query, q_size);
  convertFloatToBFloat16(key, bf16_key, kv_size);
  convertFloatToBFloat16(value, bf16_value, kv_size);

  // Split batch over number of streams (i.e. one kernel per batch)
  int num_streams = batch_size;
  hipStream_t *streams = new hipStream_t[num_streams];

  // Initialize streams
  for (int i = 0; i < num_streams; i++) {
    hipStreamCreate(&streams[i]);
  }

  // Multi-Query Attention (K,V pair shared across all heads in Q)
  size_t kv_batch_stride = seq_len * head_dim;
  size_t q_batch_stride = kv_batch_stride * num_heads;
  // Output shape [B, T, H, D]
  size_t o_batch_stride = q_batch_stride;

  // Launch kernel grids
  for (int i = 0; i < num_streams; i++) {
    // Offsets into input host arrays
    int kv_offset = kv_batch_stride * i;
    int q_offset = q_batch_stride * i;
    int o_offset = o_batch_stride * i;

    // Copy memory to device for current stream
    hipMemcpyAsync(d_q + q_offset, bf16_query + q_offset, q_offset * sizeof(__hip_bfloat16), hipMemcpyHostToDevice,
                    streams[i]);
    hipMemcpyAsync(d_k + kv_offset, bf16_key + kv_offset, kv_offset * sizeof(__hip_bfloat16), hipMemcpyHostToDevice,
                    streams[i]);
    hipMemcpyAsync(d_v + kv_offset, bf16_value + kv_offset, kv_offset * sizeof(__hip_bfloat16), hipMemcpyHostToDevice,
                    streams[i]);

    // Launch kernel for current stream
    launch_mqa_kernel(d_q, d_k, d_v, d_o, seq_len, num_heads, head_dim, block_indices, block_counts, block_size,
                      scale_factor, streams[i]);

    // Copy result back to host
    hipMemcpyAsync(output + o_offset, d_o + o_offset, o_size * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
  }

  // Wait for all streams to finish
  for (int i = 0; i < num_streams; ++i) {
    hipStreamSynchronize(streams[i]);
  }

  // Free resources
  for (int i = 0; i < num_streams; ++i) {
    hipStreamDestroy(streams[i]);
  }

  // Free device memory
  hipFree(d_q);
  hipFree(d_k);
  hipFree(d_v);
  hipFree(d_o);
}
