#include "hip/hip_runtime.h"
#include "../include/nsa.h"

using namespace nvcuda;

/**
 * Cooperative tile loading for shared memory optimization
 * Loads matrix tiles from global memory to shared memory
 */
template <typename T, int TILE_SIZE>
__device__ __inline__ void load_shared_tile(const T *global_ptr, T *shared_ptr, int global_stride, int shared_stride,
                                            int row_offset, int col_offset) {
#pragma unroll
  for (int i = threadIdx.x; i < TILE_SIZE; i += blockDim.x) {
    shared_ptr[i * shared_stride] = global_ptr[(row_offset + i) * global_stride + (col_offset)];
  }
  __syncthreads();
}

/**
 * Warp-level matrix multiply accumulate using Tensor Cores
 * Performs C = A * B where dimensions are matrix_a[M][K], matrix_b[K][N]
 *
 * Template parameters:
 * - M: Rows in matrix A and matrix D
 * - N: Columns in matrix B and matrix D
 * - K: Columns in matrix A / Rows in matrix B
 * --A: Accumulate
 *
 * All matrices must be aligned to 16-element boundaries
 */
template <int M, int N, int K, bool A>
__device__ __inline__ void bf16_warp_mma(const __hip_bfloat16 *matrix_a, // [M][K] column-major
                                         const __hip_bfloat16 *matrix_b, // [K][N] row-major
                                         float *matrix_c                // [M][N] row-major
) {
  assert(M % WMMA_M == 0);
  assert(N % WMMA_N == 0);

  int tid = threadIdx.x + threadIdx.y * blockDim.x;
  int warp_id = tid / warpSize;

  // tile output matrix
  int rows = N / WMMA_N;
  int cols = M / WMMA_M;

  // define warps tile
  int row_id = warp_id / cols;
  int col_id = warp_id % cols;

  // strides
  const int a_stride = M; // stride between cols
  const int b_stride = N; // stride between rows
  const int c_stride = N; // stride between rows

  // initial offsets
  int c_offset = row_id * (c_stride * WMMA_M) + col_id * WMMA_N;
  int a_offset = row_id * WMMA_M;
  int b_offset = col_id * WMMA_N;

  // declare wmma fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __hip_bfloat16, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __hip_bfloat16, wmma::row_major> b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> ab_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  // fill output tile with 0
  wmma::fill_fragment(ab_frag, 0.0f);

  if (warp_id < rows * cols) {
    // loop over inner dimension
#pragma unroll
    for (int k = 0; k < K; k += WMMA_K) {
      // load fragments
      wmma::load_matrix_sync(a_frag, matrix_a + a_offset + k * a_stride, a_stride);
      wmma::load_matrix_sync(b_frag, matrix_b + b_offset + k * b_stride, b_stride);

      // matmul on current tiles, AB = A * B
      wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
    }

    // optional accumulate C = AB + C
    if (A) {
      // load C into fragment
      wmma::load_matrix_sync(c_frag, matrix_c + c_offset, c_stride, wmma::mem_row_major);

      // add AB to C
      for (int i = 0; i < c_frag.num_elements; i++) {
        c_frag.x[i] = ab_frag.x[i] + c_frag.x[i];
      }

      // store tile result back to C
      wmma::store_matrix_sync(matrix_c + c_offset, c_frag, c_stride, wmma::mem_row_major);
    } else {
      // store tile result to output matrix
      wmma::store_matrix_sync(matrix_c + c_offset, ab_frag, c_stride, wmma::mem_row_major);
    }
  }
}

/**
 * Performs a sum reduction within a warp using CUDA's warp-level primitives.
 * This function assumes that threads in the same warp are participating.
 *
 * @param val The value to sum from the current thread
 * @return The sum of values from all threads in the warp (returned to all threads)
 */
__device__ __inline__ float warpReduceSum(float val) {
  // Use warp shuffle down operations to perform reduction
  // Each step halves the number of active threads but combines values
  for (int offset = 16; offset > 0; offset /= 2) {
    val += __shfl_down_sync(0xffffffff, val, offset);
  }

  // At this point, lane 0 has the sum of all values in the warp
  return val;
}

/**
 * Find the maximum value within a warp using warp shuffle operations.
 *
 * @param val The input value from each thread
 * @return The maximum value across all threads in the block (returned to all threads)
 */
__device__ __inline__ float warpReduceMax(float val) {
  float warpMax = val;

  // Use warp shuffle down to compute max within each warp
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    float shflVal = __shfl_down_sync(0xffffffff, warpMax, offset);
    warpMax = max(warpMax, shflVal);
  }

  return warpMax;
}

// Helper function to convert float to bfloat16
void convertFloatToBFloat16(const float *src, __hip_bfloat16 *dst, size_t size) {
  for (size_t i = 0; i < size; i++) {
    dst[i] = __float2bfloat16(src[i]);
  }
}

__global__ void mqa_kernel(const __hip_bfloat16 *query, const __hip_bfloat16 *key, const __hip_bfloat16 *value,
                           float *output, int seq_len, int num_heads, int head_dim, long **block_indices,
                           long *block_counts, int block_size, float scale_factor) {
  extern __shared__ __hip_bfloat16 smem[];

  // Offsets into shared memory
  __hip_bfloat16 *p_q = smem;
  __hip_bfloat16 *p_k = p_q + num_heads * head_dim;
  __hip_bfloat16 *p_v = p_k + block_size * head_dim;
  float *s = (float *)(p_v + block_size * head_dim);
  __hip_bfloat16 *p = (__hip_bfloat16 *)(s + num_heads * block_size);

  // Outer Loop (Q)
  int grid_row = blockIdx.x;
  float *o_bos = output + (grid_row * num_heads * head_dim);

  // Load query heads to smem
  // Loops over heads
  for (int head = 0; head < num_heads; head++) {
    load_shared_tile<__hip_bfloat16, THREADS_IN_BLOCK>(query, p_q + head, 1, num_heads,
                                                      (grid_row * num_heads * head_dim) + (head * head_dim), 0);
  }

  // Array to hold M for each head
  float m_p[NUM_HEADS] = {-INFINITY};
  // Array to hold accumulator
  float acc_p[NUM_HEADS] = {0};

  // Inner Loop (KV)
  long num_blocks = block_counts[grid_row];
  for (int i = 0; i < num_blocks; i++) {
    long block_id = block_indices[grid_row][i];

    // Load KV blocks to shared memory
    for (int t = 0; t < block_size; t++) {
      load_shared_tile<__hip_bfloat16, THREADS_IN_BLOCK>(key, p_k + t, 1, block_size,
                                                        (block_id * block_size + t) * head_dim, 0);
      load_shared_tile<__hip_bfloat16, THREADS_IN_BLOCK>(value, p_v + (t * head_dim), 1, 1,
                                                        (block_id * block_size + t) * head_dim, 0);
    }
    // Compute QK^T
    bf16_warp_mma<NUM_HEADS, BLOCK_SIZE, HEAD_DIM, false>(p_q, p_k, s);
    // Intermediate row operations
    for (int row = threadIdx.x / block_size; row < num_heads; row += blockDim.x) {
      int col = threadIdx.x % block_size;
      // rowmax of S
      float max = warpReduceMax(s[row * block_size + col]);
      // max(m^-1, m)
      float m = max > m_p[row] ? max : m_p[row];
      // R = exp(m^-1 - m)
      float r = expf(m_p[row] - max);

      // P = exp(m - S)
      s[row * block_size + col] = expf(s[row * block_size + col] - max);
      // rowsum of P
      float sum = warpReduceSum(s[row * block_size + col]);
      // broadcast result to warp
      sum = __shfl_sync(0xffffffff, sum, 0);

      // l = exp(m^-1 - m) * l^-1 + rowsum(P)
      float acc = r * acc_p[row] + sum;

      // O = O * diag(R)
      for (int off = col; off < head_dim; off += block_size) {
        o_bos[row * head_dim + off] *= r;
      }
      // update intermediate values
      m_p[row] = m;
      acc_p[row] = acc;
      // cast to bfloat16
      p[row * block_size + col] = __float2bfloat16(s[row * block_size + col]);
    }
    // O = O + PV
    bf16_warp_mma<NUM_HEADS, HEAD_DIM, BLOCK_SIZE, true>(p, p_v, o_bos);
  }
  // O = O / diag(l)
  for (int row = 0; row < num_heads; row++) {
    o_bos[row * head_dim + threadIdx.x] /= acc_p[row];
  }
}

void launch_mqa_kernel(const __hip_bfloat16 *query, const __hip_bfloat16 *key, const __hip_bfloat16 *value, float *output,
                       int seq_len, int num_heads, int head_dim, long **block_indices, long *block_counts,
                       int block_size, float scale_factor, hipStream_t stream) {
  // size must be compatible with wmma tiles
  assert(num_heads % 16 == 0);
  assert(head_dim % 16 == 0);
  assert(block_size % 16 == 0);

  // the dimension connot be split between blocks
  assert(head_dim <= THREADS_IN_BLOCK);

  // Number of bytes in shared memory
  size_t qkv_mem_size = (num_heads * head_dim + 2 * (block_size * head_dim)) * sizeof(__hip_bfloat16);
  size_t s_mem_size = num_heads * block_size * sizeof(float);
  size_t p_mem_size = num_heads * block_size * sizeof(__hip_bfloat16);

  size_t sharedMem = qkv_mem_size + s_mem_size + p_mem_size;

  dim3 blockDim(THREADS_IN_BLOCK);
  dim3 gridDim(seq_len);

  mqa_kernel<<<gridDim, blockDim, sharedMem, stream>>>(query, key, value, output, seq_len, num_heads, head_dim,
                                                       block_indices, block_counts, block_size, scale_factor);
}

void native_sparse_attention(const float *query, const float *key, const float *value, float *output, int batch_size,
                             int seq_len, int num_heads, int head_dim, long **block_indices, long *block_counts,
                             int block_size, float scale_factor) {
  // Number of elements in array
  size_t q_size = batch_size * seq_len * num_heads * head_dim;
  size_t kv_size = batch_size * seq_len * head_dim;
  size_t o_size = q_size;

  // Allocate memory on device
  __hip_bfloat16 *d_q, *d_k, *d_v;
  float *d_o;
  hipMalloc(&d_q, q_size * sizeof(__hip_bfloat16));
  hipMalloc(&d_k, kv_size * sizeof(__hip_bfloat16));
  hipMalloc(&d_v, kv_size * sizeof(__hip_bfloat16));
  hipMalloc(&d_o, o_size * sizeof(float));

  // Allocate host bfloat16 arrays
  __hip_bfloat16 *bf16_query = (__hip_bfloat16 *)malloc(q_size * sizeof(__hip_bfloat16));
  __hip_bfloat16 *bf16_key = (__hip_bfloat16 *)malloc(kv_size * sizeof(__hip_bfloat16));
  __hip_bfloat16 *bf16_value = (__hip_bfloat16 *)malloc(kv_size * sizeof(__hip_bfloat16));

  // Cast host float arrays to bfloat16
  convertFloatToBFloat16(query, bf16_query, q_size);
  convertFloatToBFloat16(key, bf16_key, kv_size);
  convertFloatToBFloat16(value, bf16_value, kv_size);

  // Split batch over number of streams (i.e. one kernel per batch)
  int num_streams = batch_size;
  hipStream_t *streams = new hipStream_t[num_streams];

  // Initialize streams
  for (int i = 0; i < num_streams; i++) {
    hipStreamCreate(&streams[i]);
  }

  // Multi-Query Attention (K,V pair shared across all heads in Q)
  size_t kv_batch_stride = seq_len * head_dim;
  size_t q_batch_stride = kv_batch_stride * num_heads;
  // Output shape [B, T, H, D]
  size_t o_batch_stride = q_batch_stride;

  // Launch kernel grids
  for (int i = 0; i < num_streams; i++) {
    // Offsets into input host arrays
    int kv_offset = kv_batch_stride * i;
    int q_offset = q_batch_stride * i;
    int o_offset = o_batch_stride * i;

    // Copy memory to device for current stream
    hipMemcpyAsync(d_q + q_offset, bf16_query + q_offset, q_offset * sizeof(__hip_bfloat16), hipMemcpyHostToDevice,
                    streams[i]);
    hipMemcpyAsync(d_k + kv_offset, bf16_key + kv_offset, kv_offset * sizeof(__hip_bfloat16), hipMemcpyHostToDevice,
                    streams[i]);
    hipMemcpyAsync(d_v + kv_offset, bf16_value + kv_offset, kv_offset * sizeof(__hip_bfloat16), hipMemcpyHostToDevice,
                    streams[i]);

    // Launch kernel for current stream
    launch_mqa_kernel(d_q, d_k, d_v, d_o, seq_len, num_heads, head_dim, block_indices, block_counts, block_size,
                      scale_factor, streams[i]);

    // Copy result back to host
    hipMemcpyAsync(output + o_offset, d_o + o_offset, o_size * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
  }

  // Wait for all streams to finish
  for (int i = 0; i < num_streams; ++i) {
    hipStreamSynchronize(streams[i]);
  }

  // Free resources
  for (int i = 0; i < num_streams; ++i) {
    hipStreamDestroy(streams[i]);
  }

  // Free device memory
  hipFree(d_q);
  hipFree(d_k);
  hipFree(d_v);
  hipFree(d_o);
}
